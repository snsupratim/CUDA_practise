#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512 // Size of the vectors

// CUDA kernel for vector addition
__global__ void vectorAdd(float *A, float *B, float *C, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main()
{
    int size = N * sizeof(float);
    float *h_A, *h_B, *h_C; // Host vectors
    float *d_A, *d_B, *d_C; // Device vectors

    // Allocate memory on the host
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // Initialize the host vectors
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define the block and grid sizes
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < 10; i++)
    {
        printf("C[%d] = %f\\n", i, h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}