#include "hip/hip_runtime.h"
% % cuda
#include <stdio.h>
#include <hip/hip_runtime.h>

        // Kernel function to swap two elements without using a third variable
        __global__ void
        swapKernel(int *a, int *b)
{
    // Swap the elements using arithmetic operations (addition and subtraction)
    *a = *a + *b; // a = a + b
    *b = *a - *b; // b = (a + b) - b = a
    *a = *a - *b; // a = (a + b) - a = b
}

int main()
{
    int h_a = 5, h_b = 10; // Host variables
    int *d_a, *d_b;        // Device variables

    // Allocate memory on the device
    hipMalloc((void **)&d_a, sizeof(int));
    hipMalloc((void **)&d_b, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel (1 block, 1 thread)
    swapKernel<<<1, 1>>>(d_a, d_b);

    // Copy the result back from device to host
    hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_b, d_b, sizeof(int), hipMemcpyDeviceToHost);

    // Print the swapped values
    printf("After swapping, a = %d and b = %d\n", h_a, h_b);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
